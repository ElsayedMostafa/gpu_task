#include "hip/hip_runtime.h"
//http://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf
#include <stdint.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include <time.h>
#include <cstdint>
#include <iostream>
#include <fstream>

#include "stb_image_write.h"

using namespace std;

__global__ void gpu_gray_lut(unsigned int* hist_counts_d, uint8_t* LUT_d, int size)
{
    int myid = threadIdx.x;
    LUT_d[myid] = static_cast<uint8_t>(round(255.0 * hist_counts_d[myid] / (size)));
}
inline bool exists(const std::string &name) {
    ifstream f(name.c_str());
    return f.good();
}

void cpu_histogram_equalization_grayscale(const string &path, const string &filename) {
    if (!exists(path + filename)) {
        cout << "File Doesn't Exist";
        return;
    }
    int width, height, channels;
    int desired_channels = 1;
    unsigned int *hist_counts = new unsigned int[256];
    uint8_t *LUT = new uint8_t[256];
    unsigned int *hist_counts_d;
    uint8_t *LUT_d;
    memset(hist_counts, 0, sizeof hist_counts);
    memset(LUT, 0, sizeof LUT);

    uint8_t *gray_image = stbi_load((path + filename).c_str(), &width, &height, &channels, desired_channels);

    //!histogram
    for (int i = 0; i < width * height * desired_channels; i++)
        hist_counts[gray_image[i]]++;

/*    for (int i = 0; i < 256; i++)
        cout << hist_counts[i] << ",";
    cout << endl;*/

    //!CDF
    for (int i = 1; i < 256; i++)
        hist_counts[i] += hist_counts[i - 1];

    cout << "Final value of CDF: " << hist_counts[255] << endl;


    //cuda code
    
    hipMalloc((void **) &hist_counts_d, 256 * sizeof(int));
    hipMemcpy(hist_counts_d, hist_counts, 256 * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **) &LUT_d, 256* sizeof(int));
    dim3   DimGrid(1, 1);
    dim3   DimBlock(256, 1);
    gpu_gray_lut<<< DimGrid,DimBlock >>>(hist_counts_d, LUT_d, width * height);
    hipMemcpy(LUT, LUT_d, sizeof LUT, hipMemcpyDeviceToHost);
    hipFree(hist_counts_d);
    hipFree(LUT_d);
    //!LUT cpu
    //for (int i = 0; i < 256; i++)
    //    LUT[i] = static_cast<uint8_t>(round(255.0 * hist_counts[i] / (width * height)));

    cout << "image read : " << width << " " << height << " " << channels<<endl;

    //!from LUT
    for (int i = 0; i < width * height * desired_channels; i++)
        gray_image[i] = LUT[gray_image[i]];


    stbi_write_jpg((path + "gpu_equ_" + filename).c_str(), width, height, 1, gray_image, 1000);

    stbi_image_free(gray_image);
}

void cpu_histogram_equalization_rgb(const string &path, const string &filename) {
    if (!exists(path + filename)) {
        cout << "File Doesn't Exist";
        return;
    }
    int width, height, channels;
    int desired_channels = 3;

    int hist_counts[3][256];
    uint8_t LUT[3][256];
    
    
    memset(hist_counts, 0, sizeof hist_counts);
    memset(LUT, 0, sizeof LUT);

    uint8_t *rgb_image = stbi_load((path + filename).c_str(), &width, &height, &channels, desired_channels);

    //!histogram
    for (int i = 0; i < width * height * desired_channels; i++)
        hist_counts[i % 3][rgb_image[i]]++;


    //!CDF
    for (int i = 1; i < 256; i++)
        for (int channel = 0; channel < desired_channels; ++channel)
            hist_counts[channel][i] += hist_counts[channel][i - 1];
    
    //!LUT cpu
    for (int i = 0; i < 256; i++)
       for (int channel = 0; channel < desired_channels; ++channel)
           LUT[channel][i] = static_cast<uint8_t>(round(255.0 * hist_counts[channel][i] / (width * height)));

    cout << "image read : " << width << " " << height << " " << channels<<endl;

    //!from LUT
    for (int i = 0; i < width * height * desired_channels; i += desired_channels)
        for (int channel = 0; channel < desired_channels; ++channel)
            rgb_image[i + channel] = LUT[channel][rgb_image[i + channel]];

    stbi_write_jpg((path + "gpu_equ_" + filename).c_str(), width, height, desired_channels, rgb_image, 1000);

    stbi_image_free(rgb_image);
}

int main(int argc, char **argv) {
    cpu_histogram_equalization_grayscale("./images/", "in-grayscale.jpg");
    cpu_histogram_equalization_rgb("./images/", "in-color.jpg");
    return 0;
}



