#include "hip/hip_runtime.h"
//http://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf
#include <stdint.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include <time.h>
#include <cstdint>
#include <iostream>
#include <fstream>

#include "stb_image_write.h"

using namespace std;

#define num_threads_per_block 64


inline bool exists(const std::string &name) {
    ifstream f(name.c_str());
    return f.good();
}

// Kernel definition
// Run on GPU
__global__ void histogram_kernel_gray(int size, uint8_t *gray_image, unsigned int *hist_counts) {
    //!histogram
    /*    //!needs lock :( :(  atomicInc doesn't work well :( :(
   int index = blockIdx.x * blockDim.x + threadIdx.x;

      if (index < size){
          hist_counts[gray_image[index]]=atomicInc(&hist_counts[gray_image[index]],(unsigned int)10000000);
      }
      */
    //!histogram
    for (int i = 0; i < size; i++)
        if (gray_image[i] == threadIdx.x)
            hist_counts[threadIdx.x]++;
}

// Kernel definition
// Run on GPU
__global__ void LUT_kernel_gray(uint8_t *LUT, unsigned int *hist_counts, int size) {
    LUT[threadIdx.x] = static_cast<uint8_t>(round(255.0 * hist_counts[threadIdx.x] / (size)));
}

// Kernel definition
// Run on GPU
__global__ void LUT_replacement_gray(uint8_t *LUT, uint8_t *gray_image, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
        gray_image[index] = LUT[gray_image[index]];
}

void checkerros(string m = "") {
    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        printf("Add Kernel launch failed: %s %s\n", hipGetErrorString(cudaStatus),m.c_str());
        exit(0);
    }
}

void gpu_histogram_equalization_grayscale(const string &path, const string &filename) {
    if (!exists(path + filename)) {
        cout << "File Doesn't Exist";
        return;
    }

    int width, height, channels;
    int desired_channels = 1;

    unsigned int *hist_counts = new unsigned int[256];
    uint8_t *LUT = new uint8_t[256];

    uint8_t *gray_image = stbi_load((path + filename).c_str(), &width, &height, &channels, desired_channels);
    cout << "image read : " << width << " " << height << " " << channels<<endl;

    int image_size = width * height * desired_channels;

    //! START : histogram can be parallelized
    //! define
    uint8_t *d_gray_image;
    unsigned int *d_hist_counts;

    //!Alocate and transfer data
    hipMalloc((void **) &d_gray_image, image_size * sizeof(uint8_t));
    hipMemcpy(d_gray_image, gray_image, image_size * sizeof(uint8_t), hipMemcpyHostToDevice);

    hipMalloc((void **) &d_hist_counts, 256 * sizeof(int));

    //! Launch histogram_kernel_gray() kernel on GPU
    histogram_kernel_gray << < 1, 256 >> > (image_size, d_gray_image, d_hist_counts);

    // Check for any errors launching the kernel
    checkerros("histogram_kernel_gray");

    //! wait
    hipDeviceSynchronize();
    cout << "Done:histogram_kernel_gray"<<endl;

    //!Transfer P from device to host
    hipMemcpy(hist_counts, d_hist_counts, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    //!Free device matrices
    hipFree(d_gray_image);
    hipFree(d_hist_counts);
    //! END : histogram can be parallelized

    //! START : CDF can not be parallelized
    //!CDF
    for (int i = 1; i < 256; i++)
        hist_counts[i] += hist_counts[i - 1];

    //cout << "Final value of CDF: " << hist_counts[255] << endl;
    //! END : CDF can not be parallelized

    //! START : LUT computation can be parallelized
    //!LUT
    //! define
    uint8_t *d_LUT;
    hipMalloc((void **) &d_hist_counts, 256 * sizeof(int));
    hipMemcpy(d_hist_counts, hist_counts, 256 * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **) &d_LUT, 256 * sizeof(int));

    LUT_kernel_gray << < 1, 256 >> > (d_LUT, d_hist_counts, width * height);
    checkerros("LUT_kernel_gray");

    //! wait
    hipDeviceSynchronize();
    cout << "Done:LUT_kernel_gray"<<endl;

    //!Transfer LUT from device to host
    hipMemcpy(LUT, d_LUT, 256 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    //!Free device matrices
    //! hipFree(d_LUT); free it next
    hipFree(d_hist_counts);
    //! END : LUT computation be parallelized


    int num_blocks = ((image_size + num_threads_per_block - 1) / num_threads_per_block);
    cout << "\nnumber of blocks " << num_blocks << " each one has ("<<num_threads_per_block<<" threads)" << endl;

    hipMalloc((void **) &d_gray_image, image_size * sizeof(uint8_t));
    hipMemcpy(d_gray_image, gray_image, image_size * sizeof(uint8_t), hipMemcpyHostToDevice);

    //! START : replacement using LUT can be parallelized
    LUT_replacement_gray << < num_blocks, num_threads_per_block >> > (d_LUT, d_gray_image, image_size);
    checkerros("LUT_replacement_gray");

    //! wait
    hipDeviceSynchronize();
    cout << "Done:LUT_replacement_gray"<<endl;

    //!Transfer equalized image from device to host
    hipMemcpy(gray_image, d_gray_image, image_size * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipFree(d_LUT);
    hipFree(d_gray_image);
    //! END : replacement using LUT can be parallelized

    stbi_write_jpg((path + "gpu_equ_" + filename).c_str(), width, height, 1, gray_image, 1000);

    //! free resources
    stbi_image_free(gray_image);

    delete[] hist_counts;
    delete[] LUT;
}

// Kernel definition
// Run on GPU
__global__ void histogram_kernel_rgb(int size, uint8_t *gray_image, unsigned int *hist_counts) {
    //!histogram
    for (int i = blockIdx.x; i < size; i+=3)//!channels
        if (gray_image[i] == threadIdx.x)
            hist_counts[threadIdx.x + blockIdx.x*256]++;
}

// Kernel definition
// Run on GPU
__global__ void LUT_kernel_rgb(uint8_t *LUT, unsigned int *hist_counts, int size) {
    LUT[blockIdx.x*256+threadIdx.x] = static_cast<uint8_t>(round(255.0 * hist_counts[blockIdx.x*256+threadIdx.x] / (size)));
}

// Kernel definition
// Run on GPU
__global__ void LUT_replacement_rgb(uint8_t *LUT, uint8_t *gray_image, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
        gray_image[index] = LUT[gray_image[index]+(index%3)*256];
}


void gpu_histogram_equalization_rgb(const string &path, const string &filename) {
    if (!exists(path + filename)) {
        cout << "File Doesn't Exist";
        return;
    }
    int width, height, channels;
    int desired_channels = 3;

    unsigned int *hist_counts = new unsigned int[256*3];
    uint8_t *LUT = new uint8_t[256*3];

    uint8_t *gray_image = stbi_load((path + filename).c_str(), &width, &height, &channels, desired_channels);
    cout << "image read : " << width << " " << height << " " << channels<<endl;

    int image_size = width * height * desired_channels;
    //! START : histogram can be parallelized
    //! define
    uint8_t *d_gray_image;
    unsigned int *d_hist_counts;

    //!Alocate and transfer data
    hipMalloc((void **) &d_gray_image, image_size * sizeof(uint8_t));

    hipMemcpy(d_gray_image, gray_image, image_size * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMalloc((void **) &d_hist_counts, 3*256 * sizeof(int));

    //! Launch histogram_kernel_gray() kernel on GPU
    //int num_blocks = ((width * height * desired_channels + num_threads_per_block - 1) / num_threads_per_block);
    //cout << "\nnumber of blocks " << num_blocks << " each one has (1024 threads) " << endl;

    histogram_kernel_rgb << < 3, 256 >> > (image_size, d_gray_image, d_hist_counts);

    // Check for any errors launching the kernel
    checkerros("histogram_kernel_rgb");

    //! wait
    hipDeviceSynchronize();
    cout << "Done:histogram_kernel_rgb"<<endl;
    //!Transfer P from device to host
    hipMemcpy(hist_counts, d_hist_counts, 3*256 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    //!Free device matrices
    //hipFree(d_gray_image); later
    hipFree(d_hist_counts);
    //! END : histogram can be parallelized

    //! START : CDF can not be parallelized
    //!CDF
    for (int i = 1; i < 256; i++)
        for (int channel = 0; channel < desired_channels; ++channel)
            hist_counts[i+256*channel] += hist_counts[i - 1+256*channel];

    //cout << "Final value of CDF: " << hist_counts[255] << endl;
    //! END : CDF can not be parallelized

    //! START : LUT computation can be parallelized
    //!LUT
    //! define
    uint8_t *d_LUT;
    hipMalloc((void **) &d_hist_counts, 3*256 * sizeof(int));
    hipMemcpy(d_hist_counts, hist_counts, 3*256 * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **) &d_LUT, 3*256 * sizeof(int));

    LUT_kernel_rgb << < 3, 256 >> > (d_LUT, d_hist_counts, width * height);
    checkerros("LUT_kernel_rgb");

    //! wait
    hipDeviceSynchronize();
    cout << "Done:LUT_kernel_rgb"<<endl;

    //!Transfer LUT from device to host
    hipMemcpy(LUT, d_LUT, 3*256 * sizeof(unsigned int), hipMemcpyDeviceToHost);

    //!Free device matrices
    //! hipFree(d_LUT); free it next
    hipFree(d_hist_counts);
    //! END : LUT computation be parallelized


    int num_blocks = ((image_size + num_threads_per_block - 1) / num_threads_per_block);
    //cout << "\nnumber of blocks " << num_blocks << " each one has (1024 threads)" << endl;

    //! START : replacement using LUT can be parallelized
    LUT_replacement_rgb << < num_blocks, num_threads_per_block >> > (d_LUT, d_gray_image, image_size);
    checkerros("LUT_replacement_rgb");

    //! wait
    hipDeviceSynchronize();
    cout << "Done:LUT_replacement_rgb"<<endl;

    //!Transfer equalized image from device to host
    hipMemcpy(gray_image, d_gray_image, image_size * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipFree(d_LUT);
    hipFree(d_gray_image);
    //! END : replacement using LUT can be parallelized

    stbi_write_jpg((path + "gpu_equ_" + filename).c_str(), width, height, desired_channels, gray_image, 1000);

    //! free resources
    stbi_image_free(gray_image);

    delete[] hist_counts;
    delete[] LUT;
}

void cpu_histogram_equalization_grayscale(const string &path, const string &filename) {
    if (!exists(path + filename)) {
        cout << "File Doesn't Exist";
        return;
    }
    int width, height, channels;
    int desired_channels = 1;

    int hist_counts[256];
    uint8_t LUT[256];
    memset(hist_counts, 0, sizeof hist_counts);
    memset(LUT, 0, sizeof LUT);

    uint8_t *gray_image = stbi_load((path + filename).c_str(), &width, &height, &channels, desired_channels);
    cout << "image read : " << width << " " << height << " " << channels<<endl;

    //!histogram
    for (int i = 0; i < width * height * desired_channels; i++)
        hist_counts[gray_image[i]]++;

/*    for (int i = 0; i < 256; i++)
        cout << hist_counts[i] << ",";
    cout << endl;*/

    //!CDF
    for (int i = 1; i < 256; i++)
        hist_counts[i] += hist_counts[i - 1];

    //cout << "Final value of CDF: " << hist_counts[255] << endl;

    //!LUT
    for (int i = 0; i < 256; i++)
        LUT[i] = static_cast<uint8_t>(round(255.0 * hist_counts[i] / (width * height)));



    //!from LUT
    for (int i = 0; i < width * height * desired_channels; i++)
        gray_image[i] = LUT[gray_image[i]];


    stbi_write_jpg((path + "cpu_equ_" + filename).c_str(), width, height, 1, gray_image, 1000);

    stbi_image_free(gray_image);
}

void cpu_histogram_equalization_rgb(const string &path, const string &filename) {
    if (!exists(path + filename)) {
        cout << "File Doesn't Exist";
        return;
    }
    int width, height, channels;
    int desired_channels = 3;

    int hist_counts[3][256];
    uint8_t LUT[3][256];
    memset(hist_counts, 0, sizeof hist_counts);
    memset(LUT, 0, sizeof LUT);

    uint8_t *rgb_image = stbi_load((path + filename).c_str(), &width, &height, &channels, desired_channels);
    cout << "image read : " << width << " " << height << " " << channels<<endl;

    //!histogram
    for (int i = 0; i < width * height * desired_channels; i++)
        hist_counts[i % 3][rgb_image[i]]++;

    //!CDF
    for (int i = 1; i < 256; i++)
        for (int channel = 0; channel < desired_channels; ++channel)
            hist_counts[channel][i] += hist_counts[channel][i - 1];

    //!LUT
    for (int i = 0; i < 256; i++)
        for (int channel = 0; channel < desired_channels; ++channel)
            LUT[channel][i] = static_cast<uint8_t>(round(255.0 * hist_counts[channel][i] / (width * height)));

    //!from LUT
    for (int i = 0; i < width * height * desired_channels; i += desired_channels)
        for (int channel = 0; channel < desired_channels; ++channel)
            rgb_image[i + channel] = LUT[channel][rgb_image[i + channel]];

    stbi_write_jpg((path + "cpu_equ_" + filename).c_str(), width, height, desired_channels, rgb_image, 1000);

    stbi_image_free(rgb_image);
}

int main(int argc, char **argv) {
    cout << argv[2]<<endl;

    if (strcmp(argv[1], "gg") == 0)
        gpu_histogram_equalization_grayscale("./images/", string(argv[2]));
    if (strcmp(argv[1], "cg") == 0)
        cpu_histogram_equalization_grayscale("./images/",string(argv[2]) );
    if (strcmp(argv[1], "crgb") == 0)
        cpu_histogram_equalization_rgb("./images/", string(argv[2]));
    if (strcmp(argv[1], "grgb") == 0){
        gpu_histogram_equalization_rgb("./images/", string(argv[2]));
    }

    return 0;
}



